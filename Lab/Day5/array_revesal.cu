
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define N 200

__global__ void reverse(int *a, int *b) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  b[gridDim.x - idx - 1] = a[idx];
}

void random_ints(int *p, int n) {
  int i;
  for (i = 0; i < n; i++) {
    p[i] = rand() % 100;
  }
}

int main(void) {
  int *a, *b;                 // host copies of a, b, c
  int *dev_a, *dev_b;         // device copies of a, b, c
  int size = N * sizeof(int); // we need space for 512
                              // // integers
  int i;

  // allocate device copies of a, b, c
  hipMalloc((void **)&dev_a, size);
  hipMalloc((void **)&dev_b, size);
  a = (int *)malloc(size);
  b = (int *)malloc(size);

  random_ints(a, N);
  // copy inputs to device
  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

  // launch an add() kernel with N threads
  reverse<<<N, 1>>>(dev_a, dev_b);

  // copy device result back to host copy of c
  hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);

  for (i = 0; i < N; i++) {
    if (b[i] != a[N - 1 - i]) {
      printf("Uncorrect\n");
      break;
    }
  }
  printf("Correct\n");

  free(a);
  free(b);
  hipFree(dev_a);
  hipFree(dev_b);
  return 0;
}
