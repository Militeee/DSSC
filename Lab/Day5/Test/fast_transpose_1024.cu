
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define BDIM 1024

__global__ void fast_transpose(double *a, double *b, int N) {
  // buffer
  __shared__ double buffer[BDIM];
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  // doing the transposition on the shared memory
  buffer[threadIdx.y * blockDim.x + threadIdx.x] = a[y * N + x];
  __syncthreads();
  // copy back on global memory
  b[x * N + y] = buffer[threadIdx.y * blockDim.x + threadIdx.x];
}

// naive transpose
__global__ void transpose(double *a, double *b, int N) {
  int row = (blockIdx.x * blockDim.x + threadIdx.x) / N;
  int col = (blockIdx.x * blockDim.x + threadIdx.x) % N;
  b[col * N + row] = a[row * N + col];
}

// just randomlly fill the matrix
void random_fill(double *mat, int N) {
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      mat[i * N + j] = (double)rand() / (double)RAND_MAX * 100.;
}

// Used for error-checking
void transpose_cpu(double *a, double *b, int N) {
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++)
      b[j * N + i] = a[i * N + j];
}

// check if two matrix are equals
int is_equal(double *a, double *b, int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++)
      if (b[i * N + j] != a[i * N + j])
        return 0;
  }
  return 1;
}

void print_mat(double *a, int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      printf("%.1f ", a[i * N + j]);
    }
    printf("\n");
  }
  printf("\n");
}

int main(int argc, char *argv[]) {

  double *a, *b, *c, *d; // host copies of a, b, c
  const int N = 8192;
  double *dev_a, *dev_b, *dev_c;     // device copies of a, b, c
  int size = N * N * sizeof(double); // we need space for 512
  // Get the number of block dimensions (dim1*dim2 = number of threads)
  if (argc < 3) {
    printf("Insert the dimensions, first x, second y\n");
    return -1;
  }
  // get block dimensions from command line
  const int dim1 = atoi(argv[1]);
  const int dim2 = atoi(argv[2]);
  const int Nblocks = (N * N) / 1024;
  if (dim1 * dim2 != BDIM) {
    printf("Give rigth dimensions\n");
    return -2;
  }
  dim3 grid, block;
  block.x = dim1;
  block.y = dim2;
  grid.x = N / block.x;
  grid.y = N / block.y;

  // allocate device copies of a, b, c
  hipMalloc((void **)&dev_a, size);
  hipMalloc((void **)&dev_b, size);
  a = (double *)malloc(size);
  b = (double *)malloc(size);
  d = (double *)malloc(size);
  // fill the matrix with random numbers
  random_fill(a, N);
  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  // cuda event for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  fast_transpose<<<grid, block>>>(dev_a, dev_b, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);
  //    print_mat(b,N);
  transpose_cpu(a, d, N);
  int equal = is_equal(b, d, N);
  if (equal)
    printf("Correct fast\n");
  else
    printf("Uncorrect fast\n");
  // Bandwith for reading from matrix a + writing on matrix b
  printf("Time fast= %f\n", milliseconds);
  printf("Bandwidth fast= %f\n", N * N * 2 * 8 / milliseconds / 1e6);
  free(b);
  hipFree(dev_b);
  c = (double *)malloc(size);
  hipMalloc((void **)&dev_c, size);

  hipEventRecord(start);
  transpose<<<Nblocks, 1024>>>(dev_a, dev_c, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
  equal = is_equal(c, d, N);
  if (equal)
    printf("Correct naive\n");
  else
    printf("Uncorrect naive\n");
  printf("Time naive = %f\n", milliseconds);
  printf("Bandwidth naive= %f\n", N * N * 2 * 8 / milliseconds / 1e6);
  free(a);
  free(c);
  free(d);
  hipFree(dev_a);
  hipFree(dev_c);
  return 0;
}
