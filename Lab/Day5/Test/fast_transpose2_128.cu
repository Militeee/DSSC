
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define TDIM 32
#define RDIM 4 //number of rows in a block

__global__ void fast_transpose( double* a, double* b, int N) {
    //buffer
    __shared__ double buffer[TDIM][TDIM+1];
    int blockIdx_y = blockIdx.x;
    int blockIdx_x = (blockIdx.x+blockIdx.y)%gridDim.x; 
    int y = blockIdx_y * TDIM + threadIdx.y;
    int x = blockIdx_x * TDIM + threadIdx.x;
    // doing the transposition on the shared memory
    for (int i=0; i<TDIM; i+=RDIM) {   
    	buffer[threadIdx.y+i][threadIdx.x] = a[y*N + x+i*N];
    }
    __syncthreads(); 
    // copy back on global memory
    y = blockIdx_x * TDIM + threadIdx.y;
    x = blockIdx_y * TDIM + threadIdx.x;
    for (int i=0; i<TDIM; i+=RDIM) {     
    	b[y*N + x+i*N] = buffer[threadIdx.x] [threadIdx.y+i];
    }
}

//naive transpose
__global__ void transpose( double* a, double* b, int N) {
    int row = (blockIdx.x * blockDim.x + threadIdx.x) / N;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) % N;
    b[col * N + row] = a[row * N + col];
}

// just randomlly fill the matrix  
void random_fill(double* mat, int N){
    for(int i = 0; i < N; i++)
        for(int j = 0; j < N; j++)
	    mat[i*N+j] = (double) rand()/(double) RAND_MAX * 100.;
}


// Used for error-checking
void transpose_cpu(double* a, double* b, int N){
    for(int i = 0; i < N; i++)
        for(int j = 0; j < N; j++)
            b[j*N + i] = a[i*N + j];
}



// check if two matrix are equals
int is_equal(double* a, double* b, int N){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            if(b[i*N + j] != a[i*N + j])
                return 0;
    }
    return 1;   
}

void print_mat(double* a, int N){
    for(int i = 0; i < N; i++){
	for(int j = 0; j < N; j++){
           printf("%.1f ", a[i*N+j]); 
	} 
	printf("\n");  
    }
    printf("\n");
}



int main( int argc, char* argv[] ) {

    double *a, *b, *c, *d; // host copies of a, b, c
    const int N = 8192;
    double *dev_a, *dev_b, *dev_c; // device copies of a, b, c
    int size = N * N * sizeof( double ); 
    const int Nblocks = (N*N) / 1024;
    dim3 grid, block;
    block.x = TDIM;
    block.y = RDIM;
    grid.x = N/TDIM;
    grid.y = N/TDIM;
    
    // allocate device copies of a, b, c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    a = (double*)malloc( size );
    b = (double*)malloc( size );
    d = (double*)malloc(size);
    //fill the matrix with random numbers
    random_fill( a, N );
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    //cuda event for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
   
    hipEventRecord(start);
    fast_transpose<<<grid,block>>>(dev_a, dev_b, N  );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy( b, dev_b, size, hipMemcpyDeviceToHost);
    //print_mat(b,N);
    transpose_cpu(a,d,N);
    int equal = is_equal(b,d,N);
    if(equal) printf("Correct fast\n");
    else printf("Uncorrect fast\n");
    // Bandwith for reading from matrix a + writing on matrix b
    printf("Time fast= %f\n", milliseconds);
    printf("Bandwidth fast= %f\n", N*N*2*8/milliseconds/1e6);
    free( b );
    //print_mat(d,N);
    hipFree( dev_b );
    c = (double*) malloc(size);
    hipMalloc( (void**)&dev_c, size );
        
    hipEventRecord(start);
    transpose<<<Nblocks,1024>>>(dev_a, dev_c, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost);
    equal = is_equal(c,d,N);
    if(equal) printf("Correct naive\n");
    else printf("Uncorrect naive\n");
    printf("Time naive = %f\n", milliseconds);
    printf("Bandwidth naive= %f\n", N*N*2*8/milliseconds/1e6);
    free( a ); free( c ); free(d);
    hipFree( dev_a );
    hipFree( dev_c );
    return 0;
}









